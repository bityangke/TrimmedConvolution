
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {
	template <typename Dtype>
	__global__ void conv_constrains_weight_kernel(const int nthreads, Dtype* const weight,
		const int channel, const int sz, const int group_in, const int group_out) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index % sz;
			int th = (index / sz) % sz;
			int tc = (index / sz / sz) % channel/group_in;
			int tn = index / sz / sz / channel/group_out;
			if (tn > tc) continue;
			if (tn == tc) {
				if (th < sz / 2) continue;
				else if (th == sz / 2) {
					if (tw < sz / 2)continue;
					else weight[index] = 0;
				}
				else
					weight[index] = 0;
			}
			else
				weight[index] = 0;
		}
	}
	template <typename Dtype>
	__global__ void conv_constrains_weight_kernel_v2(const int nthreads, Dtype* const weight,
		const int channel, const int sz, const int group_in, const int group_out) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index % sz;
			int th = (index / sz) % sz;
			int tc = (index / sz / sz) % channel / group_in;
			int tn = index / sz / sz / channel / group_out;
			if (tn > tc) continue;
			if (tn == tc) {
				if (th < sz / 2) continue;
				else if (th == sz / 2) {
					if (tw <= sz / 2)continue;
					else weight[index] = 0;
				}
				else
					weight[index] = 0;
			}
			else
				weight[index] = 0;
		}
	}
__global__ void sync_conv_groups() { }
template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	
	if (constrains_>0){
		int n = this->blobs_[0]->num();
		int ch = this->blobs_[0]->channels();
		int sz = this->blobs_[0]->width();
		if (constrains_ == 1) {
			conv_constrains_weight_kernel<Dtype> << <CAFFE_GET_BLOCKS(this->blobs_[0]->count()), CAFFE_CUDA_NUM_THREADS >> >
				(this->blobs_[0]->count(), this->blobs_[0]->mutable_gpu_data(), ch, sz, group_in_, group_out_);
		}
		else {
			conv_constrains_weight_kernel_v2<Dtype> << <CAFFE_GET_BLOCKS(this->blobs_[0]->count()), CAFFE_CUDA_NUM_THREADS >> >
				(this->blobs_[0]->count(), this->blobs_[0]->mutable_gpu_data(), ch, sz, group_in_, group_out_);
		}
		
		CUDA_POST_KERNEL_CHECK;
	}
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (!conv_need_backward_){
		return;
	}
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
